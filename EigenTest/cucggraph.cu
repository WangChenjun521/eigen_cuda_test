#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample implements a conjugate gradient solver on GPU
 * using CUBLAS and CUSPARSE
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas */
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <hip/hip_cooperative_groups.h>

// Utilities and system includes
#include <hip/hip_runtime_api.h>  // helper function CUDA error checking and initialization
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples

#include "cucg.h"


#if 1


#ifndef WITH_GRAPH
#define WITH_GRAPH 1
#endif


__global__ void initVectors(float *b, float *x, int N) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  for (size_t i = gid; i < N; i += gridDim.x * blockDim.x) {
    b[i] = 1.0;
    x[i] = 0.0;
  }
}

__global__ void r1_div_x(float *r1, float *r0, float *b) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid == 0) {
    b[0] = r1[0] / r0[0];
  }
}

__global__ void a_minus(float *a, float *na) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid == 0) {
    na[0] = -(a[0]);
  }
}

int cucggraph(int N, int nz, int* I, int* J, float* val, float* b) {
    printf("CUCGGRAPH    A: %d x %d, CSR,  b: %d, nz: %d\n", N, N, N, nz);
    const float tol = CGTOL;
    const int max_iter = CGMAXITER;

    float r1;

    int *d_col, *d_row;
    float *d_val, *d_x;
    float *d_r, *d_p, *d_Ax;
    int k;
    float alpha, beta, alpham1;

    hipStream_t stream1, streamForGraph;
  
    float* x = (float*)malloc(sizeof(float) * N);
    memset(x, 0, sizeof(float) * N);


    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);

    checkCudaErrors(hipblasStatus_t);

    /* Get handle to the CUSPARSE context */
    hipsparseHandle_t cusparseHandle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&cusparseHandle);

    checkCudaErrors(cusparseStatus);

    checkCudaErrors(hipStreamCreate(&stream1));

    checkCudaErrors(hipMalloc((void **)&d_col, nz * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_row, (N + 1) * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_val, nz * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_x, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_r, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_p, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_Ax, N * sizeof(float)));

    float *d_r1, *d_r0, *d_dot, *d_a, *d_na, *d_b;
    checkCudaErrors(hipMalloc((void **)&d_r1, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_r0, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_dot, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_a, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_na, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_b, sizeof(float)));

    hipsparseMatDescr_t descr = 0;
    checkCudaErrors(hipsparseCreateMatDescr(&descr));

    checkCudaErrors(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    checkCudaErrors(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

    int numBlocks = 0, blockSize = 0;
    checkCudaErrors(
        hipOccupancyMaxPotentialBlockSize(&numBlocks, &blockSize, initVectors));

    checkCudaErrors(hipMemcpyAsync(d_col, J, nz * sizeof(int),
                                    hipMemcpyHostToDevice, stream1));
    checkCudaErrors(hipMemcpyAsync(d_row, I, (N + 1) * sizeof(int),
                                    hipMemcpyHostToDevice, stream1));
    checkCudaErrors(hipMemcpyAsync(d_val, val, nz * sizeof(float),
                                    hipMemcpyHostToDevice, stream1));

    initVectors<<<numBlocks, blockSize, 0, stream1>>>(d_r, d_x, N);

    alpha = 1.0;
    alpham1 = -1.0;
    beta = 0.0;

    checkCudaErrors(hipsparseSetStream(cusparseHandle, stream1));
    checkCudaErrors(
        hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz,
                        &alpha, descr, d_val, d_row, d_col, d_x, &beta, d_Ax));

    checkCudaErrors(hipblasSetStream(cublasHandle, stream1));
    checkCudaErrors(hipblasSaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1));

    checkCudaErrors(
        hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE));
    checkCudaErrors(hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, d_r1));

    auto t0_ = clock();

    k = 1;
    // First Iteration when k=1 starts
    checkCudaErrors(hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1));
    checkCudaErrors(
        hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz,
                        &alpha, descr, d_val, d_row, d_col, d_p, &beta, d_Ax));

    checkCudaErrors(hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, d_dot));

    r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_dot, d_a);

    checkCudaErrors(hipblasSaxpy(cublasHandle, N, d_a, d_p, 1, d_x, 1));

    a_minus<<<1, 1, 0, stream1>>>(d_a, d_na);

    checkCudaErrors(hipblasSaxpy(cublasHandle, N, d_na, d_Ax, 1, d_r, 1));

    checkCudaErrors(hipMemcpyAsync(d_r0, d_r1, sizeof(float),
                                    hipMemcpyDeviceToDevice, stream1));

    checkCudaErrors(hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, d_r1));

    checkCudaErrors(hipMemcpyAsync(&r1, d_r1, sizeof(float),
                                    hipMemcpyDeviceToHost, stream1));
    checkCudaErrors(hipStreamSynchronize(stream1));
    //printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
    // First Iteration when k=1 ends
    k++;

#if WITH_GRAPH
    hipGraph_t initGraph;
    checkCudaErrors(hipStreamCreate(&streamForGraph));
    checkCudaErrors(hipblasSetStream(cublasHandle, stream1));
    checkCudaErrors(hipsparseSetStream(cusparseHandle, stream1));
    checkCudaErrors(hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal));

    r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_r0, d_b);
    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
    checkCudaErrors(hipblasSscal(cublasHandle, N, d_b, d_p, 1));
    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST);
    checkCudaErrors(hipblasSaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1));
    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);

    checkCudaErrors(
        hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_HOST));
    checkCudaErrors(
        hipsparseScsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz,
                        &alpha, descr, d_val, d_row, d_col, d_p, &beta, d_Ax));

    checkCudaErrors(hipMemsetAsync(d_dot, 0, sizeof(float), stream1));
    checkCudaErrors(hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, d_dot));

    r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_dot, d_a);

    checkCudaErrors(hipblasSaxpy(cublasHandle, N, d_a, d_p, 1, d_x, 1));

    a_minus<<<1, 1, 0, stream1>>>(d_a, d_na);

    checkCudaErrors(hipblasSaxpy(cublasHandle, N, d_na, d_Ax, 1, d_r, 1));

    checkCudaErrors(hipMemcpyAsync(d_r0, d_r1, sizeof(float),
                                    hipMemcpyDeviceToDevice, stream1));
    checkCudaErrors(hipMemsetAsync(d_r1, 0, sizeof(float), stream1));

    checkCudaErrors(hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, d_r1));

    checkCudaErrors(hipMemcpyAsync((float *)&r1, d_r1, sizeof(float),
                                    hipMemcpyDeviceToHost, stream1));

    checkCudaErrors(hipStreamEndCapture(stream1, &initGraph));
    hipGraphExec_t graphExec;
    checkCudaErrors(hipGraphInstantiate(&graphExec, initGraph, NULL, NULL, 0));
#endif

    checkCudaErrors(hipblasSetStream(cublasHandle, stream1));
    checkCudaErrors(hipsparseSetStream(cusparseHandle, stream1));



    while (r1 > tol * tol && k <= max_iter) {
#if WITH_GRAPH
        checkCudaErrors(hipGraphLaunch(graphExec, streamForGraph));
        checkCudaErrors(hipStreamSynchronize(streamForGraph));
#else
        r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_r0, d_b);
        hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
        checkCudaErrors(hipblasSscal(cublasHandle, N, d_b, d_p, 1));

        hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST);
        checkCudaErrors(hipblasSaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1));

        checkCudaErrors(hipsparseScsrmv(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha,
            descr, d_val, d_row, d_col, d_p, &beta, d_Ax));

        hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
        checkCudaErrors(hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, d_dot));

        r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_dot, d_a);

        checkCudaErrors(hipblasSaxpy(cublasHandle, N, d_a, d_p, 1, d_x, 1));

        a_minus<<<1, 1, 0, stream1>>>(d_a, d_na);
        checkCudaErrors(hipblasSaxpy(cublasHandle, N, d_na, d_Ax, 1, d_r, 1));

        checkCudaErrors(hipMemcpyAsync(d_r0, d_r1, sizeof(float),
                                        hipMemcpyDeviceToDevice, stream1));

        checkCudaErrors(hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, d_r1));
        checkCudaErrors(hipMemcpyAsync((float *)&r1, d_r1, sizeof(float),
                                        hipMemcpyDeviceToHost, stream1));
        checkCudaErrors(hipStreamSynchronize(stream1));
#endif
        //printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        k++;
    }

    auto t0 = clock() - t0_;

    int info = 0;
    if (k > max_iter)
        info = 2;

#if WITH_GRAPH
    checkCudaErrors(hipMemcpyAsync(x, d_x, N * sizeof(float),
                                    hipMemcpyDeviceToHost, streamForGraph));
    checkCudaErrors(hipStreamSynchronize(streamForGraph));
#else
    checkCudaErrors(hipMemcpyAsync(x, d_x, N * sizeof(float),
                                    hipMemcpyDeviceToHost, stream1));
    checkCudaErrors(hipStreamSynchronize(stream1));
#endif

    float rsum, diff, err = 0.0;
    for (int i = 0; i < N; i++) {
        rsum = 0.0;
        for (int j = I[i]; j < I[i + 1]; j++)
            rsum += val[j] * x[J[j]];
        diff = fabs(rsum - b[i]);
        if (diff > err) {
            err = diff;
        }
    }

#if WITH_GRAPH
    checkCudaErrors(hipGraphExecDestroy(graphExec));
    checkCudaErrors(hipGraphDestroy(initGraph));
    checkCudaErrors(hipStreamDestroy(streamForGraph));
#endif
    checkCudaErrors(hipStreamDestroy(stream1));
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    printf("solve -> %d, iter: %d, tol: %ef, err: %ef,  [ %d ] ms\n\n", info, k, sqrt(r1), err, t0);


    hipFree(d_col);
    hipFree(d_row);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ax);

 #ifdef PRINT_X
    printf("x:\n");
    for (int i = 0; i < N; i++)
        printf("%8.4f ", x[i]);
    printf("\n\n");
 #endif
    printf("\n\n");

    return 0;
}

#endif